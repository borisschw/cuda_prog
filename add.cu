
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int i = threadIdx.x;
  y[i] = x[i] + y[i];
  // int index = blockIdx.x * blockDim.x + threadIdx.x;
  // int stride = blockDim.x * gridDim.x;
  // for (int i = index; i < n; i += stride)
  //   y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 512;
  float *x, *y;
  hipProfilerStart();
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  dim3 threadsPerBlock(N, N);

  // Run kernel on 1M elements on the GPU
  int blockSize = 512;
  // int numBlocks = (N + blockSize - 1) / blockSize;
  // std::cout << "blockSize = " << blockSize << std::endl;
  // std::cout << "numBlocks = " << numBlocks << std::endl;

  add<<<1, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  hipDeviceReset();
  hipProfilerStop();

  return 0;
}