
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>

using namespace std;

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int i = threadIdx.x;
  y[i] = x[i] + y[i];
  // int index = blockIdx.x * blockDim.x + threadIdx.x;
  // int stride = blockDim.x * gridDim.x;
  // for (int i = index; i < n; i += stride)
  //   y[i] = x[i] + y[i];
}

void addVector(int N)
{
  float *x, *y;
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  dim3 threadsPerBlock(N, N);

  // Run kernel on 1M elements on the GPU
  int blockSize = 512;
  // int numBlocks = (N + blockSize - 1) / blockSize;
  // std::cout << "blockSize = " << blockSize << std::endl;
  // std::cout << "numBlocks = " << numBlocks << std::endl;

  add<<<1, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
  {
    cout<< " " << y[i] << " " ;
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  }
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  hipDeviceReset();
}


// __global__
// void MatAdd(float A[N][N], float B[N][N], float C[N][N])
// {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     int j = blockIdx.y * blockDim.y + threadIdx.y;
//     if (i < N && j < N)
//         C[i][j] = A[i][j] + B[i][j];
// }



int main(void)
{
  hipProfilerStart();

  addVector(512);

  hipProfilerStop();

  return 0;
}